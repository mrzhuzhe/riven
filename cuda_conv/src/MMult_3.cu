#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>

// CUDA runtime
#include "helper.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

// CUDA and CUBLAS functions
#define A(i,j) a[ (j)*lda + (i) ]
#define C(i,j) c[ (j)*lda + (i) ]
#define KERNEL(i,j) kernel[ (j)*kw + (i) ]

__constant__ float c_kernel[3][3];

#define BLOCK 16
__global__ void Conv_kernel(int m,  int k,  hipTextureObject_t texObj_a, int lda, 
                                    int kw, int kh, float *kernel,                                    
                                    float *c, int ldc, int stride){    
    int i, j, w, h;
    i = blockIdx.x * BLOCK + threadIdx.x;
    j = blockIdx.y * BLOCK + threadIdx.y;
  
    float sum = 0;
    if ( i < m && j < k){     
      // column major  
      for (h = 0; h < kh; h++){ 
        for (w = 0; w < kw; w++ ){
            //sum += A( i * stride + w, j * stride + h) * c_kernel[h][w];          
            sum += tex2D<float>(texObj_a, (j * stride + h)/ (float)k, (i * stride + w)/ (float)m) * c_kernel[h][w];
        }
      } 
      C( i,j ) = sum; 
    }
}

void MY_MMult( int m,  int k,  float *a, int lda, 
                                    int kw, int kh, float *kernel,                                    
                                    float *c, int ldc, int stride )
{
  //  multi channel ? multi batch ?
  //  img2col how to do img2features how to map result back
  
  int Wo = (m - kw) / stride + 1;
  int Ho = (k - kh) / stride + 1;


  // copy A to texture memory s
  // Allocate CUDA array in device memory
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipArray_t cuArray;
  hipMallocArray(&cuArray, &channelDesc, m, k);

  // Set pitch of the source (the width in memory in bytes of the 2D array pointed
  // to by src, including padding), we dont have any padding
  const size_t spitch = m * sizeof(float);
  // Copy data located at address h_data in host memory to device memory
  hipMemcpy2DToArray(cuArray, 0, 0, a, spitch, m * sizeof(float),
                      k, hipMemcpyHostToDevice);

  // Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray;

  // Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 1;
  
  // Create texture object
  hipTextureObject_t texObj = 0;
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

  // copy A to texture memory e

  dim3 block(BLOCK, BLOCK);
  dim3 grid((Wo + BLOCK - 1) / BLOCK, (Ho + BLOCK - 1)/ BLOCK);

  
  // constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_kernel), kernel, kw * kh * sizeof(float));
  
  Conv_kernel<<<grid, block>>>(Wo, Ho, texObj, lda, kw, kh, kernel, c, lda, stride);  


  // Destroy texture object
  hipDestroyTextureObject(texObj);

  // Free device memory
  hipFreeArray(cuArray);

}