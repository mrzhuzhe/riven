#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>

// CUDA runtime
#include "helper.h"
#include <hip/hip_runtime.h>

// CUDA and CUBLAS functions
#define A(i,j) a[ (j)*lda + (i) ]
#define C(i,j) c[ (j)*lda + (i) ]
#define KERNEL(i,j) kernel[ (j)*kw + (i) ]

#define BLOCK 16
__inline__ __global__ void  Conv_kernel(int m,  int k,  float *a, int lda, 
                                    int kw, int kh, float *kernel,                                    
                                    float *c, int ldc, int stride){
        int i, j, w, h;
        i = blockIdx.x * BLOCK + threadIdx.x;
        j = blockIdx.y * BLOCK + threadIdx.y;
        float sum = 0;
        if ( i < m && j < k){
          for (w = 0; w < kw; w++ ){
            for (h = 0; h < kh; h++){              
               sum += A( i * stride + w, j * stride + h) * KERNEL(w, h);          
            }
          } 
          C( i,j ) = sum; 
        }        
}

void REF_MMult_GPU( int m,  int k,  float *a, int lda, 
                                    int kw, int kh, float *kernel,                                    
                                    float *c, int ldc, int stride )
{
  int Wo = (m - kw) / stride + 1;
  int Ho = (k - kh) / stride + 1;
  dim3 block(BLOCK, BLOCK);
  dim3 grid((Wo + BLOCK - 1) / BLOCK, (Ho + BLOCK - 1)/ BLOCK);
  Conv_kernel<<<grid, block>>>(Wo, Ho, a, lda, kw, kh, kernel, c, lda, stride);  
}