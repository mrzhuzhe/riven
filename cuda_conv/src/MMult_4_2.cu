#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>

// CUDA runtime
#include "helper.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

// CUDA and CUBLAS functions
#define A(i,j) a[ (j)*lda + (i) ]
#define C(i,j) c[ (j)*lda + (i) ]
#define KERNEL(i,j) kernel[ (j)*kw + (i) ]

__constant__ float c_kernel[3][3];

#define BLOCK 16
#define STRIDES 2

__global__ void Conv_kernel(int m,  int k,  float *a, int lda, 
                                    int kw, int kh, float *kernel,                                    
                                    float *c, int ldc, int stride){    
    int i, j, w, h;
    int s_i, s_j;
    i = (blockIdx.x * BLOCK + threadIdx.x) * STRIDES;
    j = (blockIdx.y * BLOCK + threadIdx.y) * STRIDES;
    float *a_ptr = &A( i * stride, j * stride);
    float *c_ptr = &C( i * stride, j * stride);
    
    for (int idy = 0; idy < STRIDES; idy++){
      a_ptr += idy * lda;
      c_ptr += idy * lda;
      for (int idx = 0; idx < STRIDES; idx++){
        float sum = 0;
        s_i = i + idx;
        s_j = j + idy;    
        if ( s_i < m && s_j < k){     
          // column major  
          for (h = 0; h < kh; h++){ 
            for (w = 0; w < kw; w++ ){                
                sum += *(a_ptr + idx + h * lda + w ) * c_kernel[h][w];          
            }
          } 
          *(c_ptr + idx) = sum; 
        }
      
      }   
    }     
}

void MY_MMult( int m,  int k,  float *a, int lda, 
                                    int kw, int kh, float *kernel,                                    
                                    float *c, int ldc, int stride )
{
  //  multi channel ? multi batch ?
  //  img2col how to do img2features how to map result back
  
  int Wo = (m - kw) / stride + 1;
  int Ho = (k - kh) / stride + 1;
  dim3 block(BLOCK, BLOCK);
  dim3 grid((Wo + BLOCK - 1) / BLOCK / STRIDES, (Ho + BLOCK - 1)/ BLOCK / STRIDES);

  // constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_kernel), kernel, kw * kh * sizeof(float));
  
  Conv_kernel<<<grid, block>>>(Wo, Ho, a, lda, kw, kh, kernel, c, lda, stride);  
}