#include "hip/hip_runtime.h"
// nvprof is not suppoer on RTX3090
/*
======== Warning: Skipping profiling on device 0 since profiling is not supported on devices with compute capability 7.5 or higher. Profiling features on these devices are supported in the next generation GPU profiling tool NVIDIA Nsight Compute. Refer https://developer.nvidia.com/nsight-compute for more details.
Available Metrics:
                            Name   Description
*/
//  https://developer.nvidia.com/nsight-systems

//  nsignt https://docs.nvidia.com/nsight-systems/UserGuide/index.html

/*

/usr/bin/nvcc -ccbin \
g++ -I../include \
-gencode arch=compute_35,code=sm_35 \
-gencode arch=compute_37,code=sm_37 \
-gencode arch=compute_50,code=sm_50 \
-gencode arch=compute_52,code=sm_52 \
-gencode arch=compute_60,code=sm_60 \
-gencode arch=compute_61,code=sm_61 \
-gencode arch=compute_70,code=sm_70 \
-gencode arch=compute_75,code=sm_75 \
-o outputs/sgemm \
test01.cu

*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <helper_functions.h> // for benchmark purpose

#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 16

__global__ void 
sgemm_gpu_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.f;
    for (int  i = 0; i < K; ++i){
        sum += A[row*K+i] * B[i*K+col];
    }

    C[row*M+col] = alpha * sum + beta * C[row*M + col];
}

void sgemm_gpu(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    dim3 dimBlock(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 dimGrid(M/dimBlock.x, N/dimBlock.y);
    sgemm_gpu_kernel << < dimGrid, dimBlock >> >(A, B, C, N, M, K, alpha, beta); 
}

void random_init(float *data, int size)
{
    for (int i = 0; i<size; ++i){
        data[i] = (rand() & 0xFF) / (float)RAND_MAX;
    }    
}

void performance_estimation(void(*sgemm)(const float *, const float *, float *, int, int, int, float, float),
const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int test_iterations = 100;

    StopWatchInterface *timer = 0;
    
    // 
    sgemm(A, B, C, N, M, K, alpha, beta);

    //
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    //  
    for (int i = 0; i< test_iterations; i++){
        sgemm(A, B, C, N, M, K, alpha, beta);
    }
    
    //
    sdkStopTimer(&timer);

    //
    float operation_time = sdkGetAverageTimerValue(&timer);
    float operation_time_1_epoch = operation_time / test_iterations;

    printf("Operation Time = %.4f msec\n", operation_time_1_epoch);

    sdkDeleteTimer(&timer);
}

int main()
{
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;
    int N, M, K;
    float alpha = 2.f;
    float beta = 1.f;
    N = M = K = 2048;
    
    //
    A = (float *)malloc(N * K *sizeof(float));
    B = (float *)malloc(K * M *sizeof(float));
    C = (float *)malloc(N * M *sizeof(float));

    //
    hipMalloc((void **)&d_A, N * K * sizeof(float));
    hipMalloc((void **)&d_B, K * M * sizeof(float));
    hipMalloc((void **)&d_C, N * M * sizeof(float));

    //
    random_init(A, N * K);
    random_init(B, K * M);
    random_init(C, N * M);

    //  
    hipMemcpy(d_A, A, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, A, K * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, A, N * M * sizeof(float), hipMemcpyHostToDevice);

    //
    performance_estimation(sgemm_gpu, d_A, d_B, d_C, N, M, K, alpha, beta);

    //
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    //
    free(A);
    free(B);
    free(C);

    return 0;
}