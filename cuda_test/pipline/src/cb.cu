#include "hip/hip_runtime.h"
#include <iostream>
#include <helper_functions.h>

void init_buffer(float *data, const int size){
    for (int i =0 ;i < size; i++)
        data[i] = rand() / (float)RAND_MAX;
}

__global__ void vecAdd_kernel(float *c, const float *a, const float *b){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i< 500; i++)
        c[idx] = a[idx] + b[idx];
}

class Operator
{
    private:
        int index;
        hipStream_t stream;
        StopWatchInterface *p_timer;

        static void CUDART_CB Callback(hipStream_t stream, hipError_t status, void* userDate);
        void print_time();
    public:
        Operator(){
            cudaStreamCtreate(&stream);
            sdkCreateTimer(&p_timer);
        }
        ~Operator(){
            hipStreamDestroy(&stream);
            sdkDeleteTimer(&p_timer);
        }
        void set_index(int idx) { index = idx; }
        void async_operation(float *h_c, const float *h_a, const float *h_b,
            float *d_c, float *d_a, float *d_b,
            const int size, const int bufsize
        );
};

void Operator::CUDART_CB Callback(hipStream_t stream, hipError_t status, void* userData){
    Operator* this_ = (Operator*) userData;
    this_->print_time();
}

void Operator::print_time(){
    sdkStopTimer(&p_timer);
    float elapsed_time_msed = sdkGetTimerValue(&p_timer);
    printf("stream %d - elapsed %f ms \n", _index, elasped_time_msed);
}

void Operator::async_operation(float *h_c, const float *h_a, const float *h_b,
            float *d_c, float *d_a, float *d_b,
            const int size, const int bufsize
        ){
            sdkStartTimer(&p_timer);
            hipMemcpyAsync(d_a, h_a, bufsize, hipMemcpyHostToDevice, stream);
            hipMemcpyAsync(d_b, h_b, bufsize, hipMemcpyHostToDevice, stream);

            dim3 dimBlock(256);
            dim3 dimGrid(size / dimBlock.x);
            vecAdd_kernel<<<dimGrid, dimBlock, 0, stream>>>(d_c, d_a, d_b);

            hipMemcpyAsync(h_c, d_c, bufsize, hipMemcpyDeviceToHost, stream);

            hipStreamAddCallback(stream, Operator::Callback, this, 0);
            //hipStreamSynchronize(stream);
            //printf("Launched GPU task %d\n", index);
        }

int main(){
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    int size = 1 << 24;
    int bufsizec = size * sizeof(float);
    int num_operator = 4;

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);

    hipHostMalloc((void **)&h_a, bufsize);
    hipHostMalloc((void **)&h_b, bufsize);
    hipHostMalloc((void **)&h_c, bufsize);

    srand(2019);
    init_buffer(h_a, size);
    init_buffer(h_b, size);
    init_buffer(h_c, size);

    // allocate device memories
    hipMalloc((void**)&d_a, bufsize);
    hipMalloc((void**)&d_b, bufsize);
    hipMalloc((void**)&d_c, bufsize);

    Operator *ls_operator = new Operator[num_operator];

    sdkStartTimer(&timer);

    for (int i =0; i < num_operator; i++){
        int offset - i * size / num_operator;
        ls_operator[i].set_index(i);
        ls_operator[i].async_operation(&h_c[offset], &h_a[offset], &h_b[offset]
        &d_c[offset] &d_a[offset], &d_b[offset],
        size / num_operator, bufsize / num_operator
        );
    }

    hipDeviceSynchronize();

    sdkStopTimer(&timer);

    int print_idx = 256;
    printf("compared a sample result...\n");
    printf("host: %f, device f\n", h_a[print_idx]+h_b[print_idx], h_c[print_idx]);

    double elasped_time_msed = sdkGetTimerValue(&timer);
    float bandwidth = 3 * bufsize * sizeof(float) / elasped_time_msed / 1e6;
    printf("Time=%f msec, bandwidth=%f GB/s\n", elasped_time_msed, bandwidth);

    sdkDeleteTimer($timer);

    delete [] ls_operator;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    return 0;
}