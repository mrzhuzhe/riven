/*
stream 0 - elapsed 6.178000 ms 
stream 1 - elapsed 8.134000 ms 
stream 2 - elapsed 11.395000 ms 
stream 3 - elapsed 14.647000 ms 
compared a sample result...
host: 2.000000, device 2.000000
Time=14.920000 msec, bandwidth=53.974957 GB/s
*/

/*
    // openmp
   cpu 1
    cpu 3
    cpu 2
    cpu 0
    stream 1 - elapsed 6.524000 ms 
    stream 3 - elapsed 8.972000 ms 
    stream 0 - elapsed 11.969000 ms 
    stream 2 - elapsed 15.458000 ms 
    compared a sample result...
    host: 2.000000, device 2.000000
    Time=15.693000 msec, bandwidth=51.316280 GB/s
*/

#include <iostream>
#include <helper_functions.h>
#include "common.h"
#include "omp.h"

class Operator
{
    private:
        int _index;
        hipStream_t stream;
        StopWatchInterface *p_timer;

        static void CUDART_CB Callback(hipStream_t stream, hipError_t status, void* userDate);
        void print_time();
    public:
        Operator(){
            hipStreamCreate(&stream);
            sdkCreateTimer(&p_timer);
        }
        ~Operator(){
            hipStreamDestroy(stream);
            sdkDeleteTimer(&p_timer);
        }
        void set_index(int idx) { _index = idx; }
        void async_operation(float *h_c, const float *h_a, const float *h_b,
            float *d_c, float *d_a, float *d_b,
            const int size, const int bufsize
        );
};

void Operator::CUDART_CB Callback(hipStream_t stream, hipError_t status, void* userData){
    Operator* this_ = (Operator*) userData;
    this_->print_time();
}

void Operator::print_time(){    
    sdkStopTimer(&p_timer);
    float elapsed_time_msed = sdkGetTimerValue(&p_timer);
    printf("stream %d - elapsed %f ms \n", _index, elapsed_time_msed);
}

void Operator::async_operation(float *h_c, const float *h_a, const float *h_b,
            float *d_c, float *d_a, float *d_b,
            const int size, const int bufsize
        )
{
    sdkStartTimer(&p_timer);
    hipMemcpyAsync(d_a, h_a, bufsize, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, h_b, bufsize, hipMemcpyHostToDevice, stream);

    dim3 dimBlock(256);
    dim3 dimGrid(size / dimBlock.x);
    vecAdd_kernel<<<dimGrid, dimBlock, 0, stream>>>(d_c, d_a, d_b);

    hipMemcpyAsync(h_c, d_c, bufsize, hipMemcpyDeviceToHost, stream);
    //  少了一个 stream sync 变快
    hipStreamAddCallback(stream, Operator::Callback, this, 0);
}

int main(){
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    int size = 1 << 24;
    int bufsize = size * sizeof(float);
    int num_operator = 4;

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);

    hipHostMalloc((void **)&h_a, bufsize);
    hipHostMalloc((void **)&h_b, bufsize);
    hipHostMalloc((void **)&h_c, bufsize);

    srand(2023);
    init_buffer(h_a, size);
    init_buffer(h_b, size);
    init_buffer(h_c, size);

    // allocate device memories
    hipMalloc((void**)&d_a, bufsize);
    hipMalloc((void**)&d_b, bufsize);
    hipMalloc((void**)&d_c, bufsize);

    Operator *ls_operator = new Operator[num_operator];

    sdkStartTimer(&timer);

    /*
    
    for (int i =0; i < num_operator; i++){
        int offset = i * size / num_operator;
        ls_operator[i].set_index(i);
        ls_operator[i].async_operation(&h_c[offset], &h_a[offset], &h_b[offset],
        &d_c[offset], &d_a[offset], &d_b[offset],
        size / num_operator, bufsize / num_operator
        );
    }
    */


    omp_set_num_threads(num_operator);
    #pragma omp parallel
    {  
        int i = omp_get_thread_num();
        printf("cpu %i\n", i);
        int offset = i * size / num_operator;
        ls_operator[i].set_index(i);
        ls_operator[i].async_operation(&h_c[offset], &h_a[offset], &h_b[offset],
        &d_c[offset], &d_a[offset], &d_b[offset],
        size / num_operator, bufsize / num_operator
        );
            
    }

    hipDeviceSynchronize();

    sdkStopTimer(&timer);

    int print_idx = 256;
    printf("compared a sample result...\n");
    printf("host: %f, device %f\n", h_a[print_idx]+h_b[print_idx], h_c[print_idx]);

    double elasped_time_msed = sdkGetTimerValue(&timer);
    float bandwidth = 3 * bufsize * sizeof(float) / elasped_time_msed / 1e6;
    printf("Time=%f msec, bandwidth=%f GB/s\n", elasped_time_msed, bandwidth);

    sdkDeleteTimer(&timer);

    delete [] ls_operator;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    return 0;
}