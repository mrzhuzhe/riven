#include "hip/hip_runtime.h"
#include <iostream>
#include <helper_functions.h>

void init_buffer(float *data, const int size){
    for (int i =0 ;i < size; i++)
        data[i] = rand() / (float)RAND_MAX;
}

__global__ void vecAdd_kernel(float *c, const float *a, const float *b){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i< 500; i++)
        c[idx] = a[idx] + b[idx];
}

class Operator
{
    private:
        int index;
        hipStream_t stream;
    public:
        Operator(){
            cudaStreamCtreate(&stream);
        }
        ~Operator(){
            hipStreamDestroy(&stream);
        }
        void set_index(int idx) { index = idx; }
        void async_operation(float *h_c, const float *h_a, const float *h_b,
            float *d_c, float *d_a, float *d_b,
            const int size, const int bufsize
        );
}

void Operator::async_operation(float *h_c, const float *h_a, const float *h_b,
            float *d_c, float *d_a, float *d_b,
            const int size, const int bufsize
        ){
            hipMemcpyAsync(d_a, h_a, bufsize, hipMemcpyHostToDevice, stream);
            hipMemcpyAsync(d_b, h_b, bufsize, hipMemcpyHostToDevice, stream);

            dim3 dimBlock(256);
            dim3 dimGrid(size / dimBlock.x);
            vecAdd_kernel<<<dimGrid, dimBlock, 0, stream>>>(d_c, d_a, d_b);

            hipMemcpyAsync(h_c, d_c, bufsize, hipMemcpyDeviceToHost, stream);

            hipStreamSynchronize(stream);
            printf("Launched GPU task %d\n", index);
        }

int main(){
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    int size = 1 << 24;
    int bufsizec = size * sizeof(float);
    int num_operator = 4;

    hipHostMalloc((void **)&h_a, bufsize);
    hipHostMalloc((void **)&h_b, bufsize);
    hipHostMalloc((void **)&h_c, bufsize);

    srand(2019);

    return 0;
}