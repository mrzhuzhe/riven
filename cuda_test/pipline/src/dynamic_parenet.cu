#include "hip/hip_runtime.h"
#include <iostream>

#define BUF_SIZE (1 << 10)
#define BLOCKDIM 256

__global__ void child_kernel(int *data, int seed){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    atomicAdd(&data[idx], seed);
}

__global__ void parent_kernel(int *data){
    if (threadIdx.x == 0){
        int child_size = BUF_SIZE / gridDim.x;
        child_kernel<<< child_size/BLOCKDIM, BLOCKDIM >>>(&data[child_size*blockDim.x], blockDim.x+1);
    }
    hipDeviceSynchronize();
}

int main(){
    int *data;
    int num_child = 2;

    hipMallocManaged((void**)&data, BUF_SIZE*sizeof(int));
    hipMemset(data, 0, BUF_SIZE * sizeof(int));

    parent_kernel<<<num_child, 1>>>(data);

    hipDeviceSynchronize();

    int counter = 0;
    for (int i = 0; i < BUF_SIZE; i++){
        counter += data[i];
    }

    int counter_h = 0;
    for (int i = 0; i < num_child; i++){
        counter_h += (i+1);
    }

    counter_h *= BUF_SIZE / num_child;

    if (counter_h == counter)
        printf("Correct!!\n");
    else 
        printf("Error, %d %d", counter, counter_h);

    hipFree(data);

    return 0;
}