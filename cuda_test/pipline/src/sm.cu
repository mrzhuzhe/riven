
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void single_kernel(int step){
    printf("loop %d\n", step);
}

int main(){
    int n_loop = 5;
    for (int i = 0; i < n_loop; i++){
        single_kernel<<<1, 1, 0, 0>>>(i);
    }
    hipDeviceSynchronize();

    int n_stream = 5;
    hipStream_t *ls_stream;
    ls_stream = (hipStream_t*) new hipStream_t[n_stream];

    for (int i = 0; i < n_stream; i++)
        hipStreamCreate(&ls_stream[i]);

    for (int i = 0; i < n_stream; i++){
        //*
        if (i==3)
            single_kernel<<<1, 1, 0, 0>>>(i);
        else
            single_kernel<<<1, 1, 0, ls_stream[i]>>>(i);
        //*/
        //single_kernel<<<1, 1, 0, ls_stream[i]>>>(i);
        hipStreamSynchronize(ls_stream[i]);  //  multi threads sync
    }
    hipDeviceSynchronize();

    for (int i = 0; i < n_stream; i++)
        hipStreamDestroy(ls_stream[i]);
    delete [] ls_stream;

    return 0;
}