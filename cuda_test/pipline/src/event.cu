/*
    compared a sample result...
    host: 2.000000, device 2.000000
    Time=17.972000 msec, bandwidth=44.808945 GB/s

    */
#include <iostream>
#include "common.h"
#include <helper_timer.h>

int main(){
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    int size = 1 << 24;
    int bufsize = size * sizeof(float);

    hipHostMalloc((void **)&h_a, bufsize);
    hipHostMalloc((void **)&h_b, bufsize);
    hipHostMalloc((void **)&h_c, bufsize);

    srand(2023);
    init_buffer(h_a, size);
    init_buffer(h_b, size);
    //init_buffer(h_c, size);


    // allocate device memories
    hipMalloc((void**)&d_a, bufsize);
    hipMalloc((void**)&d_b, bufsize);
    hipMalloc((void**)&d_c, bufsize);

    hipMemcpyAsync(d_a, h_a, bufsize, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_b, h_b, bufsize, hipMemcpyHostToDevice);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
       
    sdkStartTimer(&timer);
    hipEventRecord(start);

    
    dim3 dimBlock(256);
    dim3 dimGrid(size / dimBlock.x);
    vecAdd_kernel<<<dimGrid, dimBlock>>>(d_c, d_a, d_b);

    hipEventRecord(stop);  // record event on kernel finished

    hipEventSynchronize(stop); // sync based on cuda event

    sdkStopTimer(&timer);

    hipMemcpyAsync(h_c, d_c, bufsize, hipMemcpyDeviceToHost);

    int print_idx = 512;
    printf("compared a sample result...\n");
    printf("host: %f, device %f\n", h_a[print_idx]+h_b[print_idx], h_c[print_idx]);

    double elasped_time_msed = sdkGetTimerValue(&timer);
    float bandwidth = 3 * bufsize * sizeof(float) / elasped_time_msed / 1e6;
    printf("Time=%f msec, bandwidth=%f GB/s\n", elasped_time_msed, bandwidth);

    sdkDeleteTimer(&timer);


    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    return 0;
}