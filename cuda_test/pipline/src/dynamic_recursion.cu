#include "hip/hip_runtime.h"
#include <iostream>

__global__ void recursive_kernel(int *data, int block_size, int depth)
{
    if (depth > 24){
        printf("CUDA does not support more than 24 depth recursion. \n");
        return;
    }
    int x_0 = blockIdx.x * block_size;
    int idx = x_0 + threadIdx.x;
    if (threadIdx.x < block_size)
        data[idx] += depth;
    
    if (depth > 0){
        __syncthreads();
        if (threadIdx.x == 0){
            int dimBlock = max(block_size/2, 32);
            int dimGrid = block_size / dimBlock;

            printf("depth [%d], offset %d, block_idx %d, block_size %d\n", depth, x_0, blockIdx.x, block_size);

            recursive_kernel<<<dimGrid, dimBlock>>>(&data[x_0], dimBlock, depth-1);
            hipDeviceSynchronize();
        }
        __syncthreads();
    }
}

int sum_depth(int depth){
    if (depth == 1)
        return 1;
    return sum_depth(depth-1) + depth;
}

int main(){
    int *data;
    int size = 1 << 9;
    int max_depth = 3;

    hipMallocManaged((void **)&data, size*sizeof(int));

    int dimBlock = 512;
    int dimGrid = size / dimBlock;
    recursive_kernel<<< dimGrid, dimBlock >>>(data, dimBlock, max_depth);

    hipDeviceSynchronize();

    int counter = 0;
    int correct_sum = sum_depth(max_depth);
    for (int i =0; i < size; i++){
        counter += (data[i] == correct_sum)? 1 : 0 ;
    }

    printf("sum_depth: %d\n", correct_sum);
    if (counter == size)
        printf("Correct!!\n");
    else
        printf("Error!! %d %d\n", counter, size);

    hipFree(data);

    return 0;
}