#include "hip/hip_runtime.h"
#include <iostream>
#include <helper_timer.h>
#include <hip/hip_runtime_api.h>
#include "common.h"

#define BLOCK_DIM 16


__global__ void sgemm_kernel(const float *A, const float *B, float *C, int M, int N, int K, float alpha, float beta){
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    float _c = 0.f;
    for (int i =0; i < K; i++){
        _c += A[row*K +i] * B[i*K+col];
    }

    C[row*N+col] = alpha * _c + beta * C[row*N + col];
}


__global__ void sgemm_memory(const float *A, const float *B, float *C, int M, int N, int K, float alpha, float beta){
    int bid_x = blockIdx.x * blockDim.x;
    int bid_y = blockIdx.y * blockDim.y;
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;
    
    float _c = 0.f;
    __shared__ float s_tile_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float s_tile_B[BLOCK_DIM][BLOCK_DIM];
    
    for (int k = 0; k < K; k += BLOCK_DIM){
        s_tile_A[tid_y][tid_x] = A[(bid_y + tid_y)*K + tid_x + k];
        s_tile_B[tid_y][tid_x] = B[(k*BLOCK_DIM+tid_y)*N + bid_x + tid_x];

        __syncthreads();

        for (int e =0; e < BLOCK_DIM; e++){
            _c += s_tile_A[tid_y][e] * s_tile_B[e][tid_x];
        }
        __syncthreads();
    }
       
    C[(bid_y + tid_y)*N + bid_x + tid_x] = alpha * _c + beta * C[(bid_y + tid_y)*N + bid_x + tid_x];
}

void sgemm(const float *A, const float *B, float *C, int M, int N, int K, float alpha, float beta){
    for (int row = 0; row < M; row ++){
        for (int col =0; col < N; col++){
            float _c = 0.f;
            for (int e = 0; e < K; e++){
                _c += A[row*K + e] * B[e*N + col];
            }
            C[row*N+col] = alpha * _c + beta * C[row*N + col];
        }
    }
}

int main(){
    float *A, *B, *C_host, *C_gpu;
    float *d_A, *d_B, *d_C;
    int M, N, K;
    float alpha = 1.f;
    float beta = 0.f;
    int n_iter = 1;
    M = N = K = 2048;

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);

    A = (float *)malloc(M*K*sizeof(float));
    B = (float *)malloc(K*N*sizeof(float));
    C_host = (float *)malloc(M*N*sizeof(float));
    C_gpu = (float *)malloc(M*N*sizeof(float));

    hipMalloc((void **)&d_A, M*K*sizeof(float));
    hipMalloc((void **)&d_B, N*K*sizeof(float));
    hipMalloc((void **)&d_C, M*N*sizeof(float));

    random_init(A, M*K);
    random_init(B, K*N);

    sdkStartTimer(&timer);

    hipMemcpy((void **)d_A, A, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void **)d_B, B, N*K*sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
    dim3 gridDim((N + BLOCK_DIM -1)/BLOCK_DIM, (M + BLOCK_DIM -1)/BLOCK_DIM);

    hipProfilerStart();

    for (int i = 0; i < n_iter; i ++ ){
        sgemm_kernel<<< gridDim, blockDim >>>(d_A, d_B, d_C, M, N, K, alpha, beta);
    }

    for (int i = 0; i < n_iter; i ++ ){
        sgemm_memory<<< gridDim, blockDim >>>(d_A, d_B, d_C, M, N, K, alpha, beta);
    }

    hipProfilerStop();

    hipDeviceSynchronize();
    sdkStopTimer(&timer);

    hipMemcpy(C_gpu, d_C, M*N*sizeof(float), hipMemcpyDeviceToHost);

    sgemm(A, B, C_host, M, N, K, alpha, beta);

    if (value_test(C_host, C_gpu, M*N)) {
        printf("ok\n");
    } else {
        printf("bad\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C_host);
    free(C_gpu);

    return 0;
}