#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"

__global__ void scan_v1_kernel(float *d_out, float *d_inp, int length){

}

void scan_v1(float *d_out, float *d_in, int length){
    dim3 dimBlock(Block_DIM);
    dim3 dimGrid(length + BLOCK_DIM -1) / BLOCK_DIM;
    scan_v1_kernel<<< dimGrid, dimBlock >>>(d_out, d_in, length);
}

void scan_host(float *h_out, float *h_in, int length, int version){
    for (int i =0; i< length; i++){
        for (int j =0; j < length; j++){
            if (i >=j){
                h_out[i] += h_in[i-j];
            }
        }
    }
}

void scan_host_v2(float *h_out, float *h_in, int length, int version){
    for (int i =0; i< length; i++){        
        h_out[i] = h_in[i];                    
    }
    int offset = 1;
    while( offset < length){
        for (int i = 0; i < length; i++){
            int idx_a = offset * (2 *i+1) -1;
            int idx_b = offset * (2 *i+2) -1;
            if (idx_a>=0 && idx_b < length)
                h_out[idx_b] += h_out[idx_a];
        }
        offset <<= 1;
    }
    offset >>= 1;
    while (offset>0){
        for (int i = 0; i < length; i++){
            int idx_a = offset * (2 *i+2) -1;
            int idx_b = offset * (2 *i+3) -1;
            if (idx_a>=0 && idx_b < length)
                h_out[idx_b] += h_out[idx_a];
        }
        offset <<= 1;
    }
}

int main(){
    srand(2023);
    float *h_input, *h_output_host, *h_output_gpu;
    float *d_input, *d_output;
    int length = BLOCK_DIM * 2;

    h_input = (float *)malloc(sizeof(float)*length);
    h_output_host = (float *)malloc(sizeof(float)*length);
    h_output_gpu = (float *)malloc(sizeof(float)*length);

    hipMalloc((void **)&d_input, sizeof(float)*length);
    hipMalloc((void **)&d_output, sizeof(float)*length);

    generate_data(h_input, length);
    print_val(h_input, 1, "input ::");

    scan_host(h_output_host, h_input, length, 1);
    print_val(h_output_host, DEBUG_OUTPUT_NUM, "result[cpu]   ::");

    hipMemcpy(d_input, h_input, sizeof(float) * length, hipMemcpyHostToDevice);
    scan_v1(d_output, d_input, length);
    hipDeviceSynchronize();
    hipMemcpy(h_output_gpu, d_output, sizeof(float) * length, hipMemcpyDeviceToHost);
    print_val(h_output_gpu, DEBUG_OUTPUT_NUM, "result[gpu_v1]::");
    if (validation(h_output_host, h_output_gpu, length))
        printf("SUCCESS!!\n");

    
    hipMemcpy(d_input, h_input, sizeof(float) * length, hipMemcpyHostToDevice);
    scan_v2(d_output, d_input, length);
    hipDeviceSynchronize();
    hipMemcpy(h_output_gpu, d_output, sizeof(float) * length, hipMemcpyDeviceToHost);
    print_val(h_output_gpu, DEBUG_OUTPUT_NUM, "result[gpu_v2]::");
    if (validation(h_output_host, h_output_gpu, length))
        printf("SUCCESS!!\n");


    // free device memory
    hipFree(d_input);
    hipFree(d_output);

    // free host memory
    free(h_input);
    free(h_output_host);
    free(h_output_gpu);

    return 0;

}