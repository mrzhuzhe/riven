#include <iostream>
//  https://zhuanlan.zhihu.com/p/526508882

int main(){
    int batch_size_train = 256;
    int num_step_train = 1600;
    int monitoring_step = 200;

    double learning_rate = 0.02f;
    double lr_decay = 5e-5f;

    bool load_pretrain = false;
    bool file_save = false;

    int batch_size_test = 10;
    int num_step_test = 1000;

    std::cout << "MNIST training with cuDnn" << std::endl;

    std::cout << "[TRAIN]" << std::endl;

    MNIST train_data_loader = MNIST("./dataset");
    train_data_loader.train(batch_size_train, true);

    Network model;
    model.add_layer(new Dense("densel", 500));
    model.add_layer(new Activation("relu", HIPDNN_ACTIVATION_RELU));
    model.add_layer(new Dense("dense2", 10));
    model.add_layer(new SOftmax("softmax"));
    model.cuda();

    if (load_pretrain)
        model.load_pretrain();
    model.train();

    // cudaProfileStart();
    int step = 0;
    Blob<float> *train_data = train_data_loader.get_data();
    Blob<float> *train_target = train_data_loader.get_target();
    train_data_loader.get_batch();
    int tp_count = 0;
    while (step < num_step_train){
        train_data->to(cuda);
        train_target->to(cuda);

        model.forward(train_data);
        tp_count += model.get_accuracy(train_target);

        model.backward(train_target);

        learning_rate *= 1.f / (1.f + lr_decay *step);

        model.update(learning_rate);


        step = train_data_loader.next();


        if (step % monitoring_step == 0)
        {
            
        }


    }

}