#include "layer.h"

#include <random>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <math.h>
#include <algorithm>

#include <fstream>
#include <iostream>

Layer::Layer(){

}

Layer::~Layer()
{
    if (output_ != nullptr) { delete output_; output_ = nullptr; }
    if (grad_input_ != nullptr) { delete grad_input_; grad_input_ = nullptr; }
    if (weights_ != nullptr) { delete weights_; weights_ = nullptr; }
    if (biases_ != nullptr) { delete biases_; biases_ = nullptr; }
    if (grad_weights_ != nullptr) { delete grad_weights_; grad_weights_ = nullptr; }
    if (grad_biases_ != nullptr) { delete grad_biases_; grad_biases_ = nullptr; }
}

void Layer::init_weight_bias(unsigned int seed){
    
}