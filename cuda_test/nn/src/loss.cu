#include "hip/hip_runtime.h"
#include "loss.h"
#include "helper.h"

#include <hip/hip_runtime.h>

CrossEntropyLoss::CrossEntropyLoss()
{
    hipMalloc((void**)&d_loss_, sizeof(float));
}

CrossEntropyLoss::~CrossEntropyLoss()
{
    if (d_loss_ != nullptr){
        hipFree(d_loss_);
        d_loss_  = nullptr;
    }

    if (d_workspace_ != nullptr){
        hipFree(d_workspace_);
    }
}

__device__ float clip(float prediction, float ep=1e-12){
    return fmin(fmax(prediction, ep), 1.f - ep);
}

__global__ void softmax_loss_kernel(float *reduced_loss, float *predict, float *target, float *workspace, int batch_size, int num_outputs){
    int batch_idx = blockDim.x * blockIdx.x + threadIdx.x;

    extern __shared__ float s_data[];
    float loss = 0.f;

    for (int c = 0; c < num_outputs; c++){
        loss += target[batch_idx*num_outputs+c] * logf(predict[batch_idx*num_outputs+c]);        
    }
    //printf("%f %f\n", target[batch_idx*num_outputs+0], predict[batch_idx*num_outputs+0]);
    workspace[batch_idx] = -loss;
    if (blockIdx.x > 0) return;

    s_data[threadIdx.x] = 0.f;
    for (int i = 0; i < batch_size; i += blockDim.x)
    {
        s_data[threadIdx.x] += workspace[threadIdx.x + i];
    }
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride >0; stride >>=1){
        if (threadIdx.x + stride < batch_size){
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0){
        reduced_loss[blockIdx.x] = s_data[0];
    }
}

void CrossEntropyLoss::init_workspace(int batch_size){
    if (d_workspace_ == nullptr)
        hipMalloc((void**)&d_workspace_, sizeof(float)*batch_size);
}

float CrossEntropyLoss::loss(Blob<float> *predict, Blob<float> *target){
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, softmax_loss_kernel, BLOCK_DIM_1D, BLOCK_DIM_1D*sizeof(float));
    int batch_size = target->n();
    int num_outputs = target->c();

    init_workspace(batch_size);

    //std::cout << "[[ LOSS ]]" << std::endl;
    //predict->print("predict", true);
    //target->print("target", true);

    int num_blocks = min(num_blocks_per_sm * num_sms, \
        (target->size() + BLOCK_DIM_1D -1) / BLOCK_DIM_1D);
    softmax_loss_kernel<<< num_blocks , BLOCK_DIM_1D, BLOCK_DIM_1D * sizeof(float), 0 >>>
    (d_loss_, predict->cuda(), target->cuda(), d_workspace_, batch_size, num_outputs);

    hipMemcpy(&h_loss_, d_loss_, sizeof(float), hipMemcpyDeviceToHost);

    return h_loss_ / float(batch_size);
}