#include "hip/hip_runtime.h"
/*
    Time= 0.268140 msec, bandwidth= 250.275482 GB/s
    host 16777216.000000, device 16777216.000000
*/

#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include <helper_timer.h>
#include "utils.h"

#define NUM_LOAD 4

namespace cg = cooperative_groups;

template <typename group_t>
__inline__ __device__ float warp_reduce_sum(group_t group, float val)
{
    //#pragma unroll 5
    for (int offset = group.size() / 2; offset > 0; offset >>=1){        
        val += group.shfl_down(val, offset);
    }
    return val;
}

__global__ void reduction_kernel(float *data_out, float *data_in, int size){    
    
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;    
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
    
    float sum[NUM_LOAD] = { 0.f };

    for (int i = idx; i < size; i += blockDim.x * gridDim.x * NUM_LOAD){
        for (int step = 0; step < NUM_LOAD; step ++ ){
            int _cur = i + step * blockDim.x * gridDim.x;
            sum[step] += (_cur < size ) ? data_in[_cur] : 0.f;
        }        
    }
    for (int i = 1; i < NUM_LOAD; i++){
        sum[0] += sum[i];
    }
    sum[0] = warp_reduce_sum(tile32, sum[0]);

    if (tile32.thread_rank() == 0){
        atomicAdd(&data_out[0], sum[0]);
    }
}

void reduction(float *d_out, float *d_in, int size, int n_threads){
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, reduction_kernel, n_threads, n_threads*sizeof(float));
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1)/ n_threads);
    
    reduction_kernel<<<n_blocks, n_threads>>>(d_out, d_in, size);
}

void run_benchmark(void (*reduce)(float *, float *, int, int), 
float *d_outPtr, float *d_inPtr, int size){
    int num_threads = 256;
    int test_iter = 100;

    reduce(d_outPtr, d_inPtr, size, num_threads);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for (int i = 0; i < test_iter; i++){
        hipMemcpy(d_outPtr, d_inPtr, size * sizeof(float), hipMemcpyDeviceToDevice);
        reduce(d_outPtr, d_inPtr, size, num_threads);               
    }

    hipDeviceSynchronize();
    sdkStopTimer(&timer);

    double elapsed_time_msed = sdkGetTimerValue(&timer) / (float)test_iter;
    float bandwidth = size * sizeof(float ) / elapsed_time_msed / 1e6;
    printf("Time= %f msec, bandwidth= %f GB/s\n", elapsed_time_msed, bandwidth);

    sdkDeleteTimer(&timer);

}

int main(){
    float *h_inPtr;
    float *d_inPtr, *d_outPtr;

    unsigned int size = 1 << 24;
    
    float result_host, result_gpu;

    srand(2019);

    h_inPtr = (float *)malloc(size*sizeof(float));

    init_input(h_inPtr, size);

    hipMalloc((void **)&d_inPtr, size*sizeof(float));
    hipMalloc((void **)&d_outPtr, size*sizeof(float));

    hipMemcpy(d_inPtr, h_inPtr, size*sizeof(float), hipMemcpyHostToDevice);

    run_benchmark(reduction, d_outPtr, d_inPtr, size);
    hipMemcpy(&result_gpu, &d_outPtr[0], sizeof(float), hipMemcpyDeviceToHost);

    result_host = get_cpu_result(h_inPtr, size);
    printf("host %f, device %f\n", result_host, result_gpu);

    hipFree(d_outPtr);
    hipFree(d_inPtr);
    free(h_inPtr);

    return 0;
}