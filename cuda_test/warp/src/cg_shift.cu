#include "hip/hip_runtime.h"
/*
    Time= 0.245790 msec, bandwidth= 273.033325 GB/s
    host 16777216.000000, device 16777216.000000
*/

#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include <helper_timer.h>
#include "utils.h"

#define NUM_LOAD 4

//https://en.cppreference.com/w/cpp/language/namespace_alias
namespace cg = cooperative_groups;
/**
    Two warp level primitives are used here for this example
    https://devblogs.nvidia.com/faster-parallel-reductions-kepler/
    https://devblogs.nvidia.com/using-cuda-warp-level-primitives/
 */

template <typename group_t>
__inline__ __device__ float warp_reduce_sum(group_t group, float val)
{
    #pragma unroll
    for (int offset = group.size() / 2; offset > 0; offset >>=1){        
        val += group.shfl_down(val, offset);
    }
    return val;
}

__inline__ __device__ float block_reduce_sum(cg::thread_block block, float val)
{
    __shared__ float shared[32];
    int warp_idx = block.thread_index().x / warpSize;

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(block);
    val = warp_reduce_sum(tile32, val);

    if (tile32.thread_rank() == 0){
        shared[warp_idx] = val;
    }

    block.sync();

    if (warp_idx == 0){
        val = (threadIdx.x < block.group_dim().x / warpSize) ? shared[tile32.thread_rank()] : 0;
        val = warp_reduce_sum(tile32, val);
    }
    return val;
}

__global__ void reduction_kernel(float *data_out, float *data_in, int size){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;    
    cg::thread_block block = cg::this_thread_block();

    float sum[NUM_LOAD] = {0.f};
    for (int i = idx; i < size; i += block.group_dim().x * gridDim.x * NUM_LOAD){
        for (int step = 0; step < NUM_LOAD; step ++ ){
            int _cur = i + step * block.group_dim().x * gridDim.x;
            sum[step] += (_cur < size ) ? data_in[_cur] : 0.f;
        }        
    }
    for (int i = 1; i < NUM_LOAD; i++){
        sum[0] += sum[i];
    }
    sum[0] = block_reduce_sum(block, sum[0]);

    if (block.thread_index().x == 0){
        data_out[block.group_index().x] = sum[0];
    }
}

int reduction(float *d_out, float *d_in, int size, int n_threads){
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, reduction_kernel, n_threads, n_threads*sizeof(float));
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1)/ n_threads);
    
    reduction_kernel<<<n_blocks, n_threads>>>(d_out, d_in, size);
    reduction_kernel<<<1, n_threads>>>(d_out, d_in, n_blocks);
    
    return 1;
}

void run_benchmark(int (*reduce)(float *, float *, int, int), 
float *d_outPtr, float *d_inPtr, int size){
    int num_threads = 256;
    int test_iter = 100;

    reduce(d_outPtr, d_inPtr, size, num_threads);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for (int i = 0; i < test_iter; i++){
        hipMemcpy(d_outPtr, d_inPtr, size * sizeof(float), hipMemcpyDeviceToDevice);
        reduce(d_outPtr, d_outPtr, size, num_threads);             
    }

    hipDeviceSynchronize();
    sdkStopTimer(&timer);

    double elapsed_time_msed = sdkGetTimerValue(&timer) / (float)test_iter;
    float bandwidth = size * sizeof(float ) / elapsed_time_msed / 1e6;
    printf("Time= %f msec, bandwidth= %f GB/s\n", elapsed_time_msed, bandwidth);

    sdkDeleteTimer(&timer);

}


int main(){
    float *h_inPtr;
    float *d_inPtr, *d_outPtr;

    unsigned int size = 1 << 24;
    
    float result_host, result_gpu;
    //int mode = 0;

    srand(2019);

    h_inPtr = (float *)malloc(size*sizeof(float));

    init_input(h_inPtr, size);

    hipMalloc((void **)&d_inPtr, size*sizeof(float));
    hipMalloc((void **)&d_outPtr, size*sizeof(float));

    hipMemcpy(d_inPtr, h_inPtr, size*sizeof(float), hipMemcpyHostToDevice);

    run_benchmark(reduction, d_outPtr, d_inPtr, size);
    hipMemcpy(&result_gpu, &d_outPtr[0], sizeof(float), hipMemcpyDeviceToHost);

    result_host = get_cpu_result(h_inPtr, size);
    printf("host %f, device %f\n", result_host, result_gpu);

    hipFree(d_outPtr);
    hipFree(d_inPtr);
    free(h_inPtr);

    return 0;
}