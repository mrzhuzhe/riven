#include "hip/hip_runtime.h"
/*
Time= 0.524270 msec, bandwidth= 128.004395 GB/s
host 16777216.000000, device 16777216.00000
*/
#include <iostream>
#include <helper_timer.h>
#include "utils.h"

__global__ void shared_reduction_kernel(float *data_out, float *data_in, int size){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];
    //__shared__ float s_data[256];

    s_data[threadIdx.x] = (idx < size) ? data_in[idx] : 0.f;

    __syncthreads();

    for (unsigned int stride =1; stride < blockDim.x; stride *= 2 ){
        if ((idx % (stride *2)) == 0)
        //if ( (idx & (stride * 2 - 1)) == 0 )  
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        data_out[blockIdx.x] = s_data[0];
}

void shared_reduction(float *d_out, float *d_in, int n_threads, int size){
    hipMemcpy(d_out, d_in, size*sizeof(float), hipMemcpyDeviceToDevice);

    while (size > 1){
        int n_blocks =(size + n_threads -1) / n_threads;
        //shared_reduction_kernel<<<n_blocks, n_threads, n_threads*sizeof(float), 0>>>(d_out, d_in, size);  // wrong
        shared_reduction_kernel<<<n_blocks, n_threads, n_threads*sizeof(float), 0>>>(d_out, d_out, size);
        //shared_reduction_kernel<<<n_blocks, n_threads>>>(d_out, d_out, size);
        size = n_blocks;
    }
}

void run_benchmark(void (*reduce)(float *, float *, int, int), 
float *d_outPtr, float *d_inPtr, int size){
    int num_threads = 256;
    int test_iter = 100;

    reduce(d_outPtr, d_inPtr, num_threads, size);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for (int i = 0; i < test_iter; i++){
        reduce(d_outPtr, d_inPtr, num_threads, size);
    }

    hipDeviceSynchronize();
    sdkStopTimer(&timer);

    double elapsed_time_msed = sdkGetTimerValue(&timer) / (float)test_iter;
    float bandwidth = size * sizeof(float ) / elapsed_time_msed / 1e6;
    printf("Time= %f msec, bandwidth= %f GB/s\n", elapsed_time_msed, bandwidth);

    sdkDeleteTimer(&timer);

}

int main(){
    float *h_inPtr;
    float *d_inPtr, *d_outPtr;

    unsigned int size = 1 << 24;
    
    float result_host, result_gpu;

    srand(2019);

    h_inPtr = (float *)malloc(size*sizeof(float));

    init_input(h_inPtr, size);

    hipMalloc((void **)&d_inPtr, size*sizeof(float));
    hipMalloc((void **)&d_outPtr, size*sizeof(float));

    hipMemcpy(d_inPtr, h_inPtr, size*sizeof(float), hipMemcpyHostToDevice);

    run_benchmark(shared_reduction, d_outPtr, d_inPtr, size);
    hipMemcpy(&result_gpu, &d_outPtr[0], sizeof(float), hipMemcpyDeviceToHost);

    result_host = get_cpu_result(h_inPtr, size);
    printf("host %f, device %f\n", result_host, result_gpu);

    hipFree(d_outPtr);
    hipFree(d_inPtr);
    free(h_inPtr);

    return 0;
}