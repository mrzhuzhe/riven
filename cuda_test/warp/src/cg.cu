#include "hip/hip_runtime.h"
/*
    Time= 0.246580 msec, bandwidth= 272.158569 GB/s
    host 16777216.000000, device 16777216.000000
    */

//  https://developer.nvidia.com/blog/cooperative-groups/

#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include <helper_timer.h>
#include "utils.h"

#define NUM_LOAD 4

//https://en.cppreference.com/w/cpp/language/namespace_alias
namespace cg = cooperative_groups;

__global__ void reduction_kernel(float *data_out, float *data_in, int size){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    cg::thread_block block = cg::this_thread_block();

    extern __shared__ float s_data[];

    float input[NUM_LOAD] = {0.f};
    for (int i = idx; i < size; i += block.group_dim().x * gridDim.x * NUM_LOAD){
        for (int step = 0; step < NUM_LOAD; step ++ ){
            int _cur = i + step * block.group_dim().x * gridDim.x;
            input[step] += (_cur < size ) ? data_in[_cur] : 0.f;
        }        
    }
    for (int i = 1; i < NUM_LOAD; i++){
        input[0] += input[i];
    }
    s_data[threadIdx.x] = input[0];
    
    block.sync();

    for (unsigned int stride = block.group_dim().x / 2; stride > 0; stride >>= 1){
        if (block.thread_index().x < stride){
            s_data[block.thread_index().x] += s_data[block.thread_index().x + stride];
            //  __syncthreads();    
            //  block.sync();   //  Benefit of cooperative group, performance may drop but provides programming flexibility
        }
        //__synchthreads(); //  Original
        block.sync();   // Equivalent operation
    }

    if (block.thread_index().x == 0){
        data_out[block.group_index().x] = s_data[0];
    }
}

int reduction(float *d_out, float *d_in, int size, int n_threads){
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, reduction_kernel, n_threads, n_threads*sizeof(float));
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1)/ n_threads);
    
    /*
    reduction_kernel<<<n_blocks, n_threads, n_threads*sizeof(float), 0>>>(d_out, d_in, size);
    reduction_kernel<<<1, n_threads, n_threads*sizeof(float), 0>>>(d_out, d_in, n_blocks);
    */
    reduction_kernel<<<n_blocks, n_threads>>>(d_out, d_in, size);
    reduction_kernel<<<1, n_threads>>>(d_out, d_in, n_blocks);
    
    return 1;
}

void run_benchmark(int (*reduce)(float *, float *, int, int), 
float *d_outPtr, float *d_inPtr, int size){
    int num_threads = 256;
    int test_iter = 100;

    reduce(d_outPtr, d_inPtr, size, num_threads);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for (int i = 0; i < test_iter; i++){
        hipMemcpy(d_outPtr, d_inPtr, size * sizeof(float), hipMemcpyDeviceToDevice);
        reduce(d_outPtr, d_outPtr, size, num_threads);               
    }

    hipDeviceSynchronize();
    sdkStopTimer(&timer);

    double elapsed_time_msed = sdkGetTimerValue(&timer) / (float)test_iter;
    float bandwidth = size * sizeof(float ) / elapsed_time_msed / 1e6;
    printf("Time= %f msec, bandwidth= %f GB/s\n", elapsed_time_msed, bandwidth);

    sdkDeleteTimer(&timer);

}


int main(){
    float *h_inPtr;
    float *d_inPtr, *d_outPtr;

    unsigned int size = 1 << 24;
    
    float result_host, result_gpu;
    //int mode = 0;

    srand(2019);

    h_inPtr = (float *)malloc(size*sizeof(float));

    init_input(h_inPtr, size);

    hipMalloc((void **)&d_inPtr, size*sizeof(float));
    hipMalloc((void **)&d_outPtr, size*sizeof(float));

    hipMemcpy(d_inPtr, h_inPtr, size*sizeof(float), hipMemcpyHostToDevice);

    run_benchmark(reduction, d_outPtr, d_inPtr, size);
    hipMemcpy(&result_gpu, &d_outPtr[0], sizeof(float), hipMemcpyDeviceToHost);

    result_host = get_cpu_result(h_inPtr, size);
    printf("host %f, device %f\n", result_host, result_gpu);

    hipFree(d_outPtr);
    hipFree(d_inPtr);
    free(h_inPtr);

    return 0;
}