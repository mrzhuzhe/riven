#include "hip/hip_runtime.h"
/*
    Time= 0.250700 msec, bandwidth= 267.685944 GB/s
    host 16777216.000000, device 16777216.000000
 */
#include <iostream>
#include <helper_timer.h>

#define NUM_LOAD 4
#include "utils.h"



__global__ void reduction_kernel_sm(float *data_out, float *data_in, int size){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    float input = 0.f;
    for (int i = idx; i < size; i+= blockDim.x * gridDim.x){
        input += data_in[i];
    }
        
    s_data[threadIdx.x] = input;

    //printf("input %f \n", input);
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1){
        if (threadIdx.x < stride){
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0){
        data_out[blockIdx.x] = s_data[0];
        //printf("  %d %f \n", blockIdx.x, data_out[blockIdx.x]);
    }

}

int reduction(float *d_out, float *d_in, int size, int n_threads){
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, reduction_kernel_sm, n_threads, n_threads*sizeof(float));
    
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1)/ n_threads);
    
    reduction_kernel_sm<<<n_blocks, n_threads, n_threads*sizeof(float), 0>>>(d_out, d_in, size);
    reduction_kernel_sm<<<1, n_threads, n_threads*sizeof(float), 0>>>(d_out, d_in, n_blocks);
    
    /*
    float result_gpu;
    hipMemcpy(&result_gpu, &d_out[0], sizeof(float), hipMemcpyDeviceToHost);
    printf(" %f \n", result_gpu);
    */
    return 1;
}


void
run_reduction(int (*reduce)(float*, float*, int, int), 
              float *d_outPtr, float *d_inPtr, int size, int n_threads)
{
    while(size > 1) 
    {
        size = reduce(d_outPtr, d_inPtr, size, n_threads);
    }
}

void run_benchmark(int (*reduce)(float *, float *, int, int), 
float *d_outPtr, float *d_inPtr, int size){
    int num_threads = 256;
    int test_iter = 100;

    reduce(d_outPtr, d_inPtr, size, num_threads);    

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
    int _size; 
    for (int i = 0; i < test_iter; i++){
        hipMemcpy(d_outPtr, d_inPtr, size * sizeof(float), hipMemcpyDeviceToDevice);
        _size = size;   // reset size
        while (_size > 1){
            _size = reduce(d_outPtr, d_outPtr, size, num_threads);            
        }        
        
        //run_reduction(reduce, d_outPtr, d_outPtr, size, num_threads);
        /*
        float result_gpu = 0;
        hipMemcpy(&result_gpu, &d_outPtr[0], sizeof(float), hipMemcpyDeviceToHost);
        printf(" %f\n", result_gpu);
        */
    }

    hipDeviceSynchronize();

    sdkStopTimer(&timer);

    

    double elapsed_time_msed = sdkGetTimerValue(&timer) / (float)test_iter;
    float bandwidth = size * sizeof(float ) / elapsed_time_msed / 1e6;
    printf("Time= %f msec, bandwidth= %f GB/s\n", elapsed_time_msed, bandwidth);

    sdkDeleteTimer(&timer);

}

int main(){
    float *h_inPtr;
    float *d_inPtr, *d_outPtr;

    unsigned int size = 1 << 24;
    
    float result_host, result_gpu;
    //int mode = 0;

    srand(2019);

    h_inPtr = (float *)malloc(size*sizeof(float));

    init_input(h_inPtr, size);

    hipMalloc((void **)&d_inPtr, size*sizeof(float));
    hipMalloc((void **)&d_outPtr, size*sizeof(float));

    hipMemcpy(d_inPtr, h_inPtr, size*sizeof(float), hipMemcpyHostToDevice);

    run_benchmark(reduction, d_outPtr, d_inPtr, size);
    hipMemcpy(&result_gpu, &d_outPtr[0], sizeof(float), hipMemcpyDeviceToHost);

    result_host = get_cpu_result(h_inPtr, size);
    printf("host %f, device %f\n", result_host, result_gpu);

    hipFree(d_outPtr);
    hipFree(d_inPtr);
    free(h_inPtr);

    return 0;
}