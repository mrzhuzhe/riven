#include "hip/hip_runtime.h"
#include <iostream>
#include <helper_timer.h>

#define NUM_LOAD 4

__global__ void shared_reduction_kernel(float *data_out, float *data_in, int size){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    float input[NUM_LOAD] = {0.f};
    for (int i = idx; i < size; i += blockDim.x * gridDim.x * NUM_LOAD){
        for (int step = 0; step < NUM_LOAD; step ++ ){
            int _cur = i + step * blockDim.x * gridDim.x;
            input[step] += (_cur < size ) ? data_in[_cur] : 0.f;
        }        
    }
    for (int i = 1; i < NUM_LOAD; i++){
        input[0] += input[i];
    }
    s_data[threadIdx.x] = input[0];
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1){
        if (threadIdx.x < stride){
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __synchthreads();
    }

    if (threadIdx.x == 0){
        data_out[blockIdx.x] = s_data[0];
    }
}

void reduction(float *d_out, float *d_in, int size, int n_threads){
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    cudaOccupancyMaxActiveBlockPerMultiprocessor(&num_block_per_sm, shared_reduction_kernel, n_threads, n_threads*sizeof(float));
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1)/ n_threads);
    
    shared_reduction_kernel<<<n_blocks, n_threads, n_threads*sizeof(float), 0>>>(d_out, d_in, size);
    shared_reduction_kernel<<<1, n_threads, n_threads*sizeof(float), 0>>>(d_out, d_in, n_blocks);
    
    return 1;
}

void run_benchmark(void (*reduce)(float *, float *, int, int), 
float *d_outPtr, float *d_inPtr, int size){
    int num_threads = 256;
    int test_iter = 100;

    reduce(d_outPtr, d_inPtr, size, num_threads);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for (int i = 0; i < test_iter; i++){
        hipMemcpy(d_outPtr, d_inPtr, size * sizeof(float), hipMemcpyDeviceToDevice);
        while (size > 1){
            size = reduce(d_outPtr, d_inPtr, size, num_threads);
        }        
    }

    hipDeviceSynchronize();
    sdkStopTimer(&timer);

    double elapsed_time_msed = sdkGetTimerValue(&timer) / (float)test_iter;
    float bandwidth = size * sizeof(float ) / elapsed_time_msed / 1e6;
    printf("Time= %f msec, bandwidth= %f GB/s\n", elapsed_time_msed, bandwidth);

    sdkDeleteTimer(&timer);

}

void init_input(float *data, int size){
    for (int i = 0; i< size; i++){
        data[i] = (rand() & 0xFF) / (float)RAND_MAX;
    }
}

float get_cpu_result(float *data, int size){
    double result = 0.f;
    for (int i = 0; i< size; i++)
        result += data[i];
    return (float)result;
}

int main(){
    float *h_inPtr;
    float *d_inPtr, *d_outPtr;

    unsigned int size = 1 << 24;
    
    float result_host, result_gpu;
    int mode = 0;

    srand(2019);

    h_inPtr = (float *)malloc(size*sizeof(float));

    init_input(h_inPtr, size);

    hipMalloc((void **)&d_inPtr, size*sizeof(float));
    hipMalloc((void **)&d_outPtr, size*sizeof(float));

    hipMemcpy(d_inPtr, h_inPtr, size*sizeof(float), hipMemcpyHostToDevice);

    run_benchmark(reduction, d_outPtr, d_inPtr, size);
    hipMemcpy(&result_gpu, &d_outPtr[0], sizeof(float), hipMemcpyDeviceToHost);

    result_host = get_cpu_result(h_inPtr, size);
    printf("host %f, device %f\n", result_host, result_gpu);

    hipFree(d_outPtr);
    hipFree(d_inPtr);
    free(h_inPtr);

    return 0;
}