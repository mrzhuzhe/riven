#include "hip/hip_runtime.h"
#include <iostream>
#include <cooperative_group.h>
#include <helper_timer.h>

template <typename group_t>
__inline__ __device__ float warp_reduce_sum(group_t group, float val)
{
    //#pragma unroll
    for (int offset = group.size() / 2; offset > 0; offset >>=1){        
        val += group.shfl_down(val, offset);
    }
    return val;
}


__inline__ __device__ float block_reduce_sum(thread_block block,float val){
    __shared__ float shared[32];
    int warp_idx = threadIdx.x / warpSize;

    thread_block_tile<32> tile32 = tiled_partition<32>(block);
    val = warp_reduce_sum(tile32, val);

    if (tile32.thread_rank() == 0){
        shared[warp_idx] = val;

    __syncthreads();

    val = (threadIdx.x < blockDim.x / warpSize) ? shared[tile32.thread_rank()] : 0;

    if (warp_idx == 0){        
        val = warp_reduce_sum(tile32, val);
    }
    return val;
}

__global__ void reduction_kernel(float *data_out, float *data_in, int size){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;    
    thread_block block = this_thread_block();
    
    float sum[NUM_LOAD] = {0.f};
    for (int i = idx; i < size; i += blockDim.x * gridDim.x * NUM_LOAD){
        for (int step = 0; step < NUM_LOAD; step ++ ){
            int _cur = i + step * blockDim.x * gridDim.x;
            sum[step] += (_cur < size ) ? data_in[_cur] : 0.f;
        }        
    }
    for (int i = 1; i < NUM_LOAD; i++){
        sum[0] += sum[i];
    }
    sum[0] = block_reduce_sum(block, sum[0]);

    if (block.thread_rank() == 0){
        atomicAdd(&data_out[0], sum[0]);
    }
}

void reduction(float *d_out, float *d_in, int size, int n_threads){
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    cudaOccupancyMaxActiveBlockPerMultiprocessor(&num_block_per_sm, shared_reduction_kernel, n_threads, n_threads*sizeof(float));
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1)/ n_threads);
    
    reduction_kernel<<<n_blocks, n_threads>>>(d_out, d_in, size);
}

void run_benchmark(void (*reduce)(float *, float *, int, int), 
float *d_outPtr, float *d_inPtr, int size){
    int num_threads = 256;
    int test_iter = 100;

    reduce(d_outPtr, d_inPtr, size, num_threads);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for (int i = 0; i < test_iter; i++){
        hipMemcpy(d_outPtr, d_inPtr, size * sizeof(float), hipMemcpyDeviceToDevice);
        reduce(d_outPtr, d_inPtr, size, num_threads);               
    }

    hipDeviceSynchronize();
    sdkStopTimer(&timer);

    double elapsed_time_msed = sdkGetTimerValue(&timer) / (float)test_iter;
    float bandwidth = size * sizeof(float ) / elapsed_time_msed / 1e6;
    printf("Time= %f msec, bandwidth= %f GB/s\n", elapsed_time_msed, bandwidth);

    sdkDeleteTimer(&timer);

}

void init_input(float *data, int size){
    for (int i = 0; i< size; i++){
        data[i] = (rand() & 0xFF) / (float)RAND_MAX;
    }
}

float get_cpu_result(float *data, int size){
    double result = 0.f;
    for (int i = 0; i< size; i++)
        result += data[i];
    return (float)result;
}

int main(){
    float *h_inPtr;
    float *d_inPtr, *d_outPtr;

    unsigned int size = 1 << 24;
    
    float result_host, result_gpu;
    int mode = 0;

    srand(2019);

    h_inPtr = (float *)malloc(size*sizeof(float));

    init_input(h_inPtr, size);

    hipMalloc((void **)&d_inPtr, size*sizeof(float));
    hipMalloc((void **)&d_outPtr, size*sizeof(float));

    hipMemcpy(d_inPtr, h_inPtr, size*sizeof(float), hipMemcpyHostToDevice);

    run_benchmark(reduction, d_outPtr, d_inPtr, size);
    hipMemcpy(&result_gpu, &d_outPtr[0], sizeof(float), hipMemcpyDeviceToHost);

    result_host = get_cpu_result(h_inPtr, size);
    printf("host %f, device %f\n", result_host, result_gpu);

    hipFree(d_outPtr);
    hipFree(d_inPtr);
    free(h_inPtr);

    return 0;
}