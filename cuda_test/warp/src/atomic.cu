#include "hip/hip_runtime.h"
/*
    Time= 27.767040 msec, bandwidth= 2.416853 GB/s
    host 16777216.000000, device 16777216.000000
*/

#include <iostream>
#include <helper_timer.h>
#include "utils.h"

__global__ void reduction_kernel(float *data_out, float *data_in, int size){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    atomicAdd(&data_out[0], data_in[idx]);
}

void reduction(float *d_out, float *d_in, int size, int n_threads){
    int n_blocks = (size + n_threads - 1)/n_threads;
    reduction_kernel<<<n_blocks, n_threads>>>(d_out, d_in, size);    
}

void run_benchmark(void (*reduce)(float *, float *, int, int), 
float *d_outPtr, float *d_inPtr, int size){
    int num_threads = 256;
    int test_iter = 100;

    reduce(d_outPtr, d_inPtr, size, num_threads);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for (int i = 0; i < test_iter; i++){
        hipMemcpy(d_outPtr, d_inPtr, size * sizeof(float), hipMemcpyDeviceToDevice);
        reduce(d_outPtr, d_outPtr, size, num_threads);               
    }

    hipDeviceSynchronize();
    sdkStopTimer(&timer);

    double elapsed_time_msed = sdkGetTimerValue(&timer) / (float)test_iter;
    float bandwidth = size * sizeof(float ) / elapsed_time_msed / 1e6;
    printf("Time= %f msec, bandwidth= %f GB/s\n", elapsed_time_msed, bandwidth);

    sdkDeleteTimer(&timer);

}


int main(){
    float *h_inPtr;
    float *d_inPtr, *d_outPtr;

    unsigned int size = 1 << 24;
    
    float result_host, result_gpu;

    srand(2023);

    h_inPtr = (float *)malloc(size*sizeof(float));

    init_input(h_inPtr, size);

    hipMalloc((void **)&d_inPtr, size*sizeof(float));
    hipMalloc((void **)&d_outPtr, size*sizeof(float));

    hipMemcpy(d_inPtr, h_inPtr, size*sizeof(float), hipMemcpyHostToDevice);

    run_benchmark(reduction, d_outPtr, d_inPtr, size);
    hipMemcpy(&result_gpu, &d_outPtr[0], sizeof(float), hipMemcpyDeviceToHost);

    result_host = get_cpu_result(h_inPtr, size);
    printf("host %f, device %f\n", result_host, result_gpu);

    hipFree(d_outPtr);
    hipFree(d_inPtr);
    free(h_inPtr);

    return 0;
}