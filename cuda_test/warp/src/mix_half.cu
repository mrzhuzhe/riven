#include "hip/hip_runtime.h"
// https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH____HALF__ARITHMETIC.html

/*

#ifndef __CUDA_ARCH__
#define __CUDA_ARCH__ 800
#endif
*/

#include <hip/hip_fp16.h>
#include <helper_timer.h>
#include <cstdio>
#include <sm_61_intrinsics.h>
#include "cb.h"

__global__ void hfma_kernel(half *d_x, half *d_y, float *d_z, int size){
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    half2 *dual_x = reinterpret_cast<half2*>(d_x);
    half2 *dual_y = reinterpret_cast<half2*>(d_y);
    float2 *dual_z = reinterpret_cast<float2*>(d_z);

    extern __shared__ float2 s_data[];

#if __CUDA_ARCH__ >= 530
    for (int i = idx_x; i < size; i += stride){
        //dual_z[i] = __half22float2(dual_x[i], dual_y[i]);
        //  calling a __device__ function("__internal_device_float2_to_half2_rn(float, float)") from a __host__ __device__ function("__floats2half2_rn") is not allowed
        dual_z[i] = __half22float2(__hmul2(dual_x[i], dual_y[i]));
    }
#else 
    for (int i = idx_x; i < size; i += stride){
        dual_z[i] = __half22float2(dual_x[i]) * __half22float2(dual_y[i]);
    }
#endif

}


void fhma_host(half *h_x, half *h_y, float *h_z, int size){
    #pragma omp parallel
    {
    #pragma omp for
        for (int i = 0; i < size; i++){
            h_z[i] = __half2float(h_x[i]) * __half2float(h_y[i]);
        }
    }
}


int main() {

    CBuffer<half> X, Y;
    CBuffer<float> Z;

    int size = 1 << 26;
    srand(2023);

    X.init(size, true);
    Y.init(size, true);
    Z.init(size, true);

    X.cuda();
    Y.cuda();
    Z.cuda();

    int n_threads = 256;
    int num_sms;
    int num_blocks_per_sms;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sms, hfma_kernel, n_threads, n_threads*sizeof(float2));
    int n_blocks = min(num_blocks_per_sms * num_sms, (size/2 + n_threads -1) / n_threads);

    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    hfma_kernel<<< n_blocks, n_threads, n_threads * sizeof(float2) >>>(X.d_ptr_, Y.d_ptr_, Z.d_ptr_, size/2);

    hipDeviceSynchronize();
    sdkStopTimer(&timer);

    double elapsed_time_msed = sdkGetTimerValue(&timer);
    float ops = (float)size  / elapsed_time_msed * 1e6;
    printf("FMA, FLOPS = %f GFLops, Operation Time= %f msec\n", ops, elapsed_time_msed);

    fhma_host(X.h_ptr_, Y.h_ptr_, Z.h_ptr_, size);

    int diff_count = Z.diff_count();
    (diff_count == 0) ? printf("Success!!\n") : printf("Counted diff!! (%d times)\n", diff_count);

    sdkDeleteTimer(&timer);    

    return 0;
}
