
#include <hip/hip_runtime.h>
#include <iostream>
//#define STR_HELPER(x) #x
//#define STR(x) STR_HELPER(x)

__global__ void print_index_kernel(){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_idx = threadIdx.x / warpSize;
    int lane_idx = threadIdx.x & (warpSize-1);
    
    //const char my_compile_time_arch[] = STR(__CUDA_ARCH__);

    if ((lane_idx & (warpSize/2-1)) == 0)


        #if __CUDA_ARCH__ >= 880
            printf("__CUDA_ARCH__");
        #else 
            printf("else");
        #endif
        printf("final");

        //printf("__CUDA_ARCH__: %s\n", my_compile_time_arch);
        //printf(" %5d\t%5d\t %2d\t%2d\n", idx, blockIdx.x, warp_idx, lane_idx);
}


int main(){
    int gridDim = 4, blockDim = 128;
    puts("thread, block, warp, lane");
    print_index_kernel<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    return 0;
}