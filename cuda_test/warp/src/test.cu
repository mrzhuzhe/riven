//  https://zhuanlan.zhihu.com/p/572820783 warp mask test


#include <hip/hip_runtime.h>
#include <iostream>
#define N 128

// kernel 是否可以指定返回值
__global__ void test_kernel(float *a, float *b, int n){
    int tid = threadIdx.x;
    //printf("----- tid: %d\n", tid);
    if (tid > n)
        return;
    float temp = a[tid];
    printf(" %d-%f ", tid, temp);
    b[tid] = __all_sync(0xffffffff, temp > 48); 
}

int main(){
    const int n_blocks = 1;
    const int n_threads = N / n_blocks;
    const int size = N;
    const int m_size = size * sizeof(float);
    
    float *h_a, *h_b;    
    float *d_a, *d_b;
    int n;

    n = size;
    h_a = (float *)malloc(m_size);
    h_b = (float *)malloc(m_size);

    for (int i=0; i < size; i++ ){
        h_a[i] = i;
    }

    hipMalloc((void **)&d_a, m_size);
    hipMalloc((void **)&d_b, m_size);
    hipMemcpy(d_a, h_a, m_size, hipMemcpyHostToDevice);

    test_kernel<<<n_blocks, n_threads>>>(d_a, d_b, n);
    hipMemcpy(h_b, d_b, m_size, hipMemcpyDeviceToHost);
    printf("\n");

    for (int i=0; i < size; i++ ){
        printf(" %f ", h_b[i]);
        if ((i+1) % 10 == 0){
            printf("\n");
        } 
    }
    printf("\n");
    hipDeviceSynchronize();

    return 0;
}