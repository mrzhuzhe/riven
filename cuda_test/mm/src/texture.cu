
#include <hip/hip_runtime.h>
#include <iostream>

// Simple transformation kernel
__global__ void transformKernel(float* output,
                                hipTextureObject_t texObj,
                                int width, int height,
                                float theta)
{
    // Calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = x / (float)width;
    float v = y / (float)height;

    

    // Transform coordinates
    ///*
    u -= 0.5f;
    v -= 0.5f;
    float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
    float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;
    
    if (y == 1 && x == 1){
        //printf(" %d %d %f \n", x, y , tex2D<float>(texObj, (x / (float)width), (y / (float)height)));
        printf(" %d %d %f %f %f \n", x, y , u , v, tex2D<float>(texObj, x, y));
        //std::cout << tex2D<float>(texObj, tu, tv) << std::endl;
    }
    
    // Read from texture and write to global memory
    output[y * width + x] = tex2D<float>(texObj, tu, tv);
    //*/
}

// Host code
int main()
{
    const int height = 1024;
    const int width = 1024;
    float angle = 0.5;

    // Allocate and set some host data
    float *h_data = (float *)std::malloc(sizeof(float) * width * height);
    for (int i = 0; i < height * width; ++i)
        h_data[i] = i;
    printf(" %f \n", h_data[100]);
    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray_t cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);

    // Set pitch of the source (the width in memory in bytes of the 2D array pointed
    // to by src, including padding), we dont have any padding
    const size_t spitch = width * sizeof(float);
    // Copy data located at address h_data in host memory to device memory
    hipMemcpy2DToArray(cuArray, 0, 0, h_data, spitch, width * sizeof(float),
                        height, hipMemcpyHostToDevice);

    
    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    // Allocate result of transformation in device memory
    float *output;
    hipMalloc(&output, width * height * sizeof(float));

    // Invoke kernel
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
                    (height + threadsperBlock.y - 1) / threadsperBlock.y);
    transformKernel<<<numBlocks, threadsperBlock>>>(output, texObj, width, height,
                                                    angle);
    // Copy data from device back to host
    hipMemcpy(h_data, output, width * height * sizeof(float),
                hipMemcpyDeviceToHost);
    
    printf(" %f \n", h_data[1023*1024]);

    // Destroy texture object
    hipDestroyTextureObject(texObj);

    // Free device memory
    hipFreeArray(cuArray);
    hipFree(output);

    // Free host memory
    free(h_data);

    return 0;
}