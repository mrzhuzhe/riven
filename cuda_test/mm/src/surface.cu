
#include <hip/hip_runtime.h>
#include <iostream>
// Simple copy kernel
__global__ void copyKernel(hipSurfaceObject_t inputSurfObj,
                           hipSurfaceObject_t outputSurfObj,
                           int width, int height)
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        uchar4 data;
        // Read from input surface
        surf2Dread(&data,  inputSurfObj, x * 4, y);
        if ( x == 0 && y < 10){
            printf(" %d ", data);
        }
        // Write to output surface
        surf2Dwrite(data, outputSurfObj, x * 4, y);
    }
}

// Host code
int main()
{
    const int height = 1024;
    const int width = 1024;

    // Allocate and set some host data
    unsigned char *h_data =
        (unsigned char *)std::malloc(sizeof(unsigned char) * width * height * 4);
    for (int i = 0; i < height * width * 4; ++i)
        h_data[i] = i;

    // Allocate CUDA arrays in device memory
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    hipArray_t cuInputArray;
    hipMallocArray(&cuInputArray, &channelDesc, width, height,
                    hipArraySurfaceLoadStore);
    hipArray_t cuOutputArray;
    hipMallocArray(&cuOutputArray, &channelDesc, width, height,
                    hipArraySurfaceLoadStore);

    // Set pitch of the source (the width in memory in bytes of the 2D array
    // pointed to by src, including padding), we dont have any padding
    const size_t spitch = 4 * width * sizeof(unsigned char);
    // Copy data located at address h_data in host memory to device memory
    hipMemcpy2DToArray(cuInputArray, 0, 0, h_data, spitch,
                        4 * width * sizeof(unsigned char), height,
                        hipMemcpyHostToDevice);

    // Specify surface
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    // Create the surface objects
    resDesc.res.array.array = cuInputArray;
    hipSurfaceObject_t inputSurfObj = 0;
    hipCreateSurfaceObject(&inputSurfObj, &resDesc);
    resDesc.res.array.array = cuOutputArray;
    hipSurfaceObject_t outputSurfObj = 0;
    hipCreateSurfaceObject(&outputSurfObj, &resDesc);

    // Invoke kernel
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
                    (height + threadsperBlock.y - 1) / threadsperBlock.y);
    copyKernel<<<numBlocks, threadsperBlock>>>(inputSurfObj, outputSurfObj, width,
                                                height);

    // Copy data from device back to host
    hipMemcpy2DFromArray(h_data, spitch, cuOutputArray, 0, 0,
                            4 * width * sizeof(unsigned char), height,
                            hipMemcpyDeviceToHost);

    // Destroy surface objects
    hipDestroySurfaceObject(inputSurfObj);
    hipDestroySurfaceObject(outputSurfObj);

    // Free device memory
    hipFreeArray(cuInputArray);
    hipFreeArray(cuOutputArray);

    // Free host memory
    free(h_data);

  return 0;
}