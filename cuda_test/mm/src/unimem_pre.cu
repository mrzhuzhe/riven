#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

__global__ void init(int n, float *x, float *y){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i+= stride ){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
}

__global__ void add(int n, float *x, float *y){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i+= stride ){
        y[i] = x[i] + y[i];
    }
}

int main(){
    printf("hello pinmem\n");
    int N = 1 << 20;
    float *x, *y;
    int device = -1;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));


    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    init<<<numBlocks, blockSize>>>(N, x, y);

    hipGetDevice(&device);
    hipMemPrefetchAsync(x, N*sizeof(float), device, NULL);
    hipMemPrefetchAsync(y, N*sizeof(float), device, NULL);

    add<<<numBlocks, blockSize>>>(N, x, y);

    hipMemPrefetchAsync(y, N*sizeof(float), hipCpuDeviceId, NULL);

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "\n Max error " << maxError << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}