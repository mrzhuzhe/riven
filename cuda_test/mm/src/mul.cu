#include "hip/hip_runtime.h"
//  ======== Warning: nvprof is not supported on devices with compute capability 8.0 and higher.
//                 Use NVIDIA Nsight Systems for GPU tracing and CPU sampling and NVIDIA Nsight Compute for GPU profiling.
//                  Refer https://developer.nvidia.com/tools-overview for more details.

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <helper_functions.h> // for benchmark purpose

#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 16

__global__ void sgemm_gpu_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{    
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.f;
    for (int  i = 0; i < K; ++i){
        sum += A[row*K+i] * B[i*K+col];
    }

    C[row*M+col] = alpha * sum + beta * C[row*M + col];
}

void sgemm_gpu(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    dim3 dimBlock(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 dimGrid(M/dimBlock.x, N/dimBlock.y);
    sgemm_gpu_kernel<<<dimGrid, dimBlock>>>(A, B, C, N, M, K, alpha, beta); 
}


void performance_estimation(void(*sgemm)(const float *, const float *, float *, int, int, int, float, float),
const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int test_iterations = 100;

    StopWatchInterface *timer = 0;
    
    // 
    sgemm(A, B, C, N, M, K, alpha, beta);

    //
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    //  
    for (int i = 0; i< test_iterations; i++){
        sgemm(A, B, C, N, M, K, alpha, beta);
    }
    
    //
    sdkStopTimer(&timer);

    //
    float operation_time = sdkGetAverageTimerValue(&timer);
    float operation_time_1_epoch = operation_time / test_iterations;

    printf("Operation Time = %.4f msec\n", operation_time_1_epoch);

    sdkDeleteTimer(&timer);
}


void random_init(float *data, int size)
{
    for (int i = 0; i<size; ++i){
        data[i] = (float)i;
        //data[i] = 1.f;
        //data[i] = (rand() & 0xFF) / (float)RAND_MAX;       
    }    
}

void print_output(float *a, float *b, float *c, int m, int n, int k) {
    int count = 10;
    int begin = 100;
    for (int idx=begin;idx<begin+count;idx++){
        int col = idx % n; // mod
        int row = idx / n; // residual
        printf("%f =", c[idx]);
        for (int i = 0; i < k; i++){
            if (i > 0)
                printf(" +");
            printf(" %f * %f", a[row*k+i], b[i*k+col]);
        }
        printf("\n");
    }
}



int main()
{
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;
    int N, M, K;
    float alpha = 1.f;
    float beta = 0.f;
    //  N = M = K = 2048 * 2048;    // out of range
    N = M = K = 128; 
    
    //
    A = (float *)malloc(N * K *sizeof(float));
    B = (float *)malloc(K * M *sizeof(float));
    C = (float *)malloc(N * M *sizeof(float));

    //
    hipMalloc((void **)&d_A, N * K * sizeof(float));
    hipMalloc((void **)&d_B, K * M * sizeof(float));
    hipMalloc((void **)&d_C, N * M * sizeof(float));

    //    
    random_init(A, N * K);
    random_init(B, K * M);
    //random_init(C, N * M);

       
    hipMemcpy(d_A, A, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * M * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(d_C, C, N * M * sizeof(float), hipMemcpyHostToDevice);

    //
    performance_estimation(sgemm_gpu, d_A, d_B, d_C, N, M, K, alpha, beta);

    hipMemcpy(C, d_C, N * M *sizeof(float), hipMemcpyDeviceToHost);

    //print_output(A, B, C, M, N, K);    
    
    //
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    //
    free(A);
    free(B);
    free(C);

    return 0;
}