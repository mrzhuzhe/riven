
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ void init(int n, float *x, float *y){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i+= stride ){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
}

__global__ void add(int n, float *x, float *y){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i+= stride ){
        y[i] = x[i] + y[i];
    }
}

int main(){
    printf("hello pinmem\n");
    int N = 1 << 20;
    float *x, *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));


    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    init<<<numBlocks, blockSize>>>(N, x, y);
    add<<<numBlocks, blockSize>>>(N, x, y);

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "\n Max error " << maxError << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}