#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>

struct DisableCopy {
    DisableCopy() = default;
    DisableCopy(DisableCopy const &) = delete;
    DisableCopy &operator=(DisableCopy const &) = delete;
}

template <class T>
struct  CudaArray : DisableCopy
{
    hipArray *m_cuArray{};
    uint3 m_dim{};
    explicit CudaArray(uint3 const &_dim): m_dim(_dim){
        hipExtent extent = make_hipExtent(m_dim.x, m_dim.y, m_dim.z);
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<T>();
        hipMalloc3DArray(&m_cuArray, &channelDesc, extent, hipArraySurfaceLoadStore);
    }

    void copyIn(T const *_data){
        hipMemcpy3DParms copy3DParams{};
        copy3DParams.srcPtr = make_hipPitchedPtr((void *)_data, m_dim.x* sizeof(T), m_dim.x, m_dim.y);
        copy3DParams.dstArray = m_cuArray;
        copy3DParams.extent = make_hipExtent(m_dim.x, m_dim.y, m_dim.z);
        copy3Dparams.kind = hipMemcpyHostToDevice;
        hipMemcpy3D(&copy3DParams);
    }

    void copyOut(T *_data){
        hipMemcpy3DParms copy3DParams{};
        copy3DParams.srcArray = m_cuArray;
        copy3DParams.dstPtr = make_hipPitchedPtr((void *)_data, m_dim.x * sizeof(T), m_dim.x, m_dim.y);
        copy3DParams.extent = make_hipExtent(m_dim.x, m_dim.y, m_dim.z);
        copy3Dparams.kind = hipMemcpyDeviceToHost;
        hipMemcpy3D(&copy3DParams);
    }

    hipArray *getArray() const {
        return m_cuArray;
    }

    ~CudaArray(){
        hipFreeArray(m_cuArray);
    }

};

template <class T>
struct CudaSurfaceAccessor {
    hipSurfaceObject_t m_cuSuf;
    template <hipSurfaceBoundaryMode mode = hipBoundaryModeTrap>
    __device__ __forceinline__ T read(int x, int y, int z) const {
        return surf3Dread<T>(m_cuSuf, x*sizeof(T), y, z, mode);
    }
    template <hipSurfaceBoundaryMode mode = hipBoundaryModeTrap>
    __device__ __forceinline__ T write(int x, int y, int z) const {
        return surf3Dwrite<T>(val, m_cuSuf, x*sizeof(T), y, z, mode);
    }
};

template <class T>
struct  CudaSurface : CudaArray<T>
{
    hipSurfaceObject_t m_cuSur{};
    explicit CudaSurface(uint3 const &_dim): CudaArray<T>(_dim){
        hipResourceDesc resDesc{};
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = CudaArray<T>::getArray();
        hipCreateSurfaceObject(&m_cuSuf, &resDesc);
    }  

    hipSurfaceObject_t getSurface() const {
        return m_cuSuf;
    }

    CudaSurfaceAccessor<T> accessSurface() const {
        return {m_cuSuf}
    }
    ~CudaSurface(){
        cudaDEstroySurfaceObject(m_cuSuf);
    }

};


int main(){
   
    return 0;
}