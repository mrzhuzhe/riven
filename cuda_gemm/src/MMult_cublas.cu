#include <assert.h>
#include <stdlib.h>

// CUDA runtime
#include "helper.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

// CUDA and CUBLAS functions

void MY_MMult(hipblasHandle_t handle, int m, int n, int k, float *d_A, int lda,
              float *d_B, int ldb, float *d_C, int ldc) {

  const float alpha = 1.0f;
  const float beta = 0.0f;

  checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha,
                              d_B, n, d_A, k, &beta, d_C, n));
}