#include "hip/hip_runtime.h"
#include <assert.h>
#include <cmath>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <stdlib.h>
#include <vector>

// CUDA runtime
#include "helper.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define SMEM_LDA (128)
#define SMEM_LDB (128)

// remove original guard
__device__ __forceinline__ void ldg32_nc_0(float &reg, const void *ptr) {
  asm volatile("{.reg .pred p;\n"
               "mov.b32 %0, 0;\n"
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDACC_VER_MINOR__ >= 4 &&                 \
    __CUDA_ARCH__ >= 750
               "ld.global.nc.L2::128B.f32 %0, [%1];}\n"
#else
               "ld.global.nc.f32 %0, [%1];}\n"
#endif
               : "=f"(reg)
               : "l"(ptr));
}

__device__ __forceinline__ uint32_t smem_u32addr(const void *smem_ptr) {
  uint32_t addr;
  asm("{.reg .u64 u64addr;\n"
      " cvta.to.shared.u64 u64addr, %1;\n"
      " cvt.u32.u64 %0, u64addr;}\n"
      : "=r"(addr)
      : "l"(smem_ptr));

  return addr;
}

__device__ __forceinline__ void lds128(float &reg0, float &reg1, float &reg2,
                                       float &reg3, const uint32_t &addr) {
  asm volatile("ld.shared.v4.f32 {%0, %1, %2, %3}, [%4];\n"
               : "=f"(reg0), "=f"(reg1), "=f"(reg2), "=f"(reg3)
               : "r"(addr));
}

__device__ __forceinline__ void sts128(const float &reg0, const float &reg1,
                                       const float &reg2, const float &reg3,
                                       const uint32_t &addr) {
  asm volatile("st.shared.v4.f32 [%0], {%1, %2, %3, %4};\n"
               :
               : "r"(addr), "f"(reg0), "f"(reg1), "f"(reg2), "f"(reg3));
}

__device__ __forceinline__ void sts32(const float &reg, const uint32_t &addr) {
  asm volatile("st.shared.f32 [%0], %1;\n" : : "r"(addr), "f"(reg));
}

// 256 threads  perblock, 2 blocks per multiprocessor
/**
 * version 10 相对于 version 9 的特点是
 * 1. 用了 uint32_t 代替 64bit 的 smem 地址，然后用 lds128 来加载数据，3080
 * 上实跑没卵用。
 * 2. gmem 到 smem 时，用 reg 做搬运, 3080/Tesla T4  实跑都没有卵用。
 */
__global__ __launch_bounds__(256, 2) void sgemm_128x128x8(int m, int n, int k,
                                                          const float *a,
                                                          const float *b,
                                                          float *c) {

  __shared__ __align__(
      16 * 1024) char smem[24 * 1024]; // 16KB shared memory for buffer
  float *ashare = reinterpret_cast<float *>(smem);
  float *bshare =
      reinterpret_cast<float *>(smem + 16 * 1024); // 8k shared mem for B
  float sum[8][8] = {0};
  float panelA[8] = {0}, panelB[8] = {0};

  int from_a = (blockIdx.y * 128 + threadIdx.x / 8 * 4) * k + threadIdx.x % 8;
  int from_b = (threadIdx.x / 32) * n + blockIdx.x * 128 + threadIdx.x % 32;

  float a_ldg_reg[4], b_ldg_reg[4];

  uint32_t a_sts_addr = smem_u32addr(ashare + (threadIdx.x % 8) * SMEM_LDA +
                                     (threadIdx.x / 8) * 4);
  uint32_t b_sts_addr =
      smem_u32addr(bshare + (threadIdx.x / 32) * SMEM_LDB + (threadIdx.x % 32));

  uint32_t aptr_base = smem_u32addr(ashare + (threadIdx.x / 16) * 4);
  uint32_t bptr_base = smem_u32addr(bshare + (threadIdx.x % 16) * 4);

  for (int loop = 0; loop < k; loop += 8) {
// part1: gmem to smem
// load gmem to smem for ashare
#pragma unroll
    for (int i = 0; i < 4; ++i) {
      ldg32_nc_0(a_ldg_reg[i],
                 (const char *)(a + from_a) + i * k * sizeof(float));
    }
    sts128(a_ldg_reg[0], a_ldg_reg[1], a_ldg_reg[2], a_ldg_reg[3], a_sts_addr);

// load gmem to smem for bshare
#pragma unroll
    for (int i = 0; i < 4; ++i) {
      ldg32_nc_0(b_ldg_reg[i],
                 (const char *)(b + from_b) + i * 32 * sizeof(float));
    }
#pragma unroll
    for (int i = 0; i < 4; ++i) {
      sts32(b_ldg_reg[i], b_sts_addr + i * 32 * sizeof(float));
    }

    __syncthreads();
    from_a += 8;
    from_b += 8 * n;

// part2: calculation
// 计算 2x2 个 4x4
#pragma unroll
    for (int subk = 0; subk < 8; ++subk) {
      lds128(panelA[0], panelA[1], panelA[2], panelA[3],
             aptr_base + (subk * SMEM_LDA) * sizeof(float));
      lds128(panelA[4], panelA[5], panelA[6], panelA[7],
             aptr_base + (subk * SMEM_LDA + 64) * sizeof(float));

      lds128(panelB[0], panelB[1], panelB[2], panelB[3],
             bptr_base + (subk * SMEM_LDB) * sizeof(float));
      lds128(panelB[4], panelB[5], panelB[6], panelB[7],
             bptr_base + (subk * SMEM_LDB + 64) * sizeof(float));

#pragma unroll
      for (int i = 0; i < 8; ++i) {
#pragma unroll
        for (int j = 0; j < 8; ++j) {
          sum[i][j] += panelA[i] * panelB[j];
        }
      }
    }
    __syncthreads();
  }

  // part3: save to C
  int write_offset = (blockIdx.y * 128 + (threadIdx.x / 16) * 4) * n +
                     blockIdx.x * 128 + (threadIdx.x % 16) * 4;
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    for (int j = 0; j < 4; ++j) {
      c[write_offset + i * n + j] = sum[i][j];
      c[write_offset + i * n + j + 64] = sum[i][j + 4];
      c[write_offset + (i + 64) * n + j] = sum[i + 4][j];
      c[write_offset + (i + 64) * n + j + 64] = sum[i + 4][j + 4];
    }
  }
}

#undef SMEM_LDA
#undef SMEM_LDB

void MY_MMult(hipblasHandle_t handle, int m, int n, int k, float *d_A, int lda,
              float *d_B, int ldb, float *d_C, int ldc) {

  constexpr int BLOCK = 128;
  dim3 grid((m + BLOCK - 1) / BLOCK, (n + BLOCK - 1) / BLOCK);

  sgemm_128x128x8<<<grid, 256>>>(m, n, k, d_A, d_B, d_C);
}