#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>

// CUDA runtime
#include "helper.h"
//#include <hipblas.h>
#include <hip/hip_runtime.h>

/**
 * vertical Y horizon X
 */
template <int BLOCK>
__global__ void sgemm(int m, int n, int k, float *a, int lda, float *b, int ldb,
                      float *c, int ldc) {
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int bx = blockIdx.x;
  const int by = blockIdx.y;

  float *abeg = a + by * BLOCK * k;
  float *bbeg = b + bx * BLOCK;
  float *end_a = abeg + k;

  float sum = 0.f;

  for (float *a_ptr = abeg, *b_ptr = bbeg; a_ptr < end_a; a_ptr += BLOCK, b_ptr += BLOCK * n){
    __shared__ float ashare[BLOCK][BLOCK];
    __shared__ float bshare[BLOCK][BLOCK];

    ashare[ty][tx] = a_ptr[ty*k + tx];
    bshare[ty][tx] = b_ptr[ty*n + tx];
    __syncthreads();

    for (int kk = 0; kk < BLOCK; ++kk){
      sum += ashare[ty][kk] * bshare[kk][tx];
    }
    __syncthreads();
  }

  c[(BLOCK*by + ty) * n + BLOCK * bx + tx] = sum; 
}

void MY_MMult(hipblasHandle_t handle, int m, int n, int k, float *d_A, int lda,
              float *d_B, int ldb, float *d_C, int ldc) {

  constexpr int BLOCK = 16;
  // subm, subn, subk
  dim3 block(BLOCK, BLOCK);
  dim3 grid((m + BLOCK - 1) / BLOCK, (n + BLOCK - 1) / BLOCK);

  sgemm<BLOCK><<<grid, block>>>(m, n, k, d_A, lda, d_B, ldb, d_C, ldc);
}