#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>

// CUDA runtime
#include "helper.h"
//#include <hipblas.h>
#include <hip/hip_runtime.h>

/**
 * naive 实现 vertical X horizon Y
 */
template <int BLOCK>
__global__ void sgemm(int m, int n, int k, float *a, int lda, float *b, int ldb,
                      float *c, int ldc) {
  int _m = blockIdx.x * BLOCK + threadIdx.x;
  int _n = blockIdx.y * BLOCK + threadIdx.y;
  if (_m < m and _n < n) {
    float sum = 0.f;
    for (int i = 0; i < k; ++i) {
      sum += a[_m * k + i] * b[i * n + _n];
    }
    c[_m * n + _n] = sum;
  }
}

void MY_MMult(hipblasHandle_t handle, int m, int n, int k, float *d_A, int lda,
              float *d_B, int ldb, float *d_C, int ldc) {

  constexpr int BLOCK = 16;
  // subm, subn, subk
  dim3 block(BLOCK, BLOCK);
  dim3 grid((m + BLOCK - 1) / BLOCK, (n + BLOCK - 1) / BLOCK);

  sgemm<BLOCK><<<grid, block>>>(m, n, k, d_A, lda, d_B, ldb, d_C, ldc);
}